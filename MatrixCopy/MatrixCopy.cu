#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void copy_matrix_kernel(const float* A, float* B, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N * N) {
        B[idx] = A[idx];
    }
}

// A, B are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* A, float* B, int N) {
    int total = N * N;
    int threadsPerBlock = 256;
    int blocksPerGrid = (total + threadsPerBlock - 1) / threadsPerBlock;
    copy_matrix_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, N);
    hipDeviceSynchronize();
} 