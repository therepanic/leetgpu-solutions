#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void relu_kernel(const float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        if (input[idx] < 0) {
            output[idx] = 0;
        } else {
            output[idx] = input[idx];  
        }
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}
